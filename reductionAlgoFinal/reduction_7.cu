#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

template<unsigned int blockSize>
__device__ int* reduction_7(int *g_idata, int *g_odata)
{
    static __shared__ int sdata[THREADS];
    // each thread loads one element from global to shared mem
    //unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[threadIdx.x] = 0;
    while (i < CUDASIZE)
    {
        sdata[threadIdx.x] += g_idata[i] + g_idata[i+blockSize];
        i += gridSize;
    }
    __syncthreads();
    // do reduction in shared mem
    if (blockSize >= 512)
    {
        if (threadIdx.x < 256)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (threadIdx.x < 128)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (threadIdx.x < 64)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 64];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32)
        warpReduce<blockSize>(sdata, threadIdx.x);
    // write result for this block to global mem
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    //implement second reduction for the summed array
    __syncthreads();
    i = blockIdx.x*(blockSize*2) + threadIdx.x;
    gridSize = blockSize*2*gridDim.x;
    sdata[threadIdx.x] = 0;
    while (i < CUDASIZE)
    {
        sdata[threadIdx.x] += g_odata[i] + g_odata[i+blockSize];
        i += gridSize;
    }
    __syncthreads();
    if (blockSize >= 512)
    {
        if (threadIdx.x < 256)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (threadIdx.x < 128)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (threadIdx.x < 64)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 64];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32) warpReduce<blockSize>(sdata, threadIdx.x);
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    return g_odata;
}
template<unsigned int blockSize>
__device__ int* reduction_71(int *g_idata, int *g_odata)
{
    static __shared__ int sdata[THREADS];
    // each thread loads one element from global to shared mem
    //unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    unsigned int gridSize = blockDim.x*2*gridDim.x;
    sdata[threadIdx.x] = 0;
    while (i < CUDASIZE)
    {
        sdata[threadIdx.x] += g_idata[i] + g_idata[i+blockDim.x];
        i += gridSize;
    }
    __syncthreads();
    // do reduction in shared mem
    if (blockSize >= 1024)
    {
        if (threadIdx.x < 512)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 512];
        }
        __syncthreads();
    }
    if (blockSize >= 512)
    {
        if (threadIdx.x < 256)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (threadIdx.x < 128)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (threadIdx.x < 64)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 64];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32)
        warpReduce<blockSize>(sdata, threadIdx.x);
    // write result for this block to global mem
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    return g_odata;
}

template<unsigned int blockSize>
__device__ int* reduction_72(int *g_idata, int *g_odata)
{
    static __shared__ int sdata[BLOCKS/4];
    // each thread loads one element from global to shared mem
    //unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    unsigned int gridSize = blockDim.x*2*gridDim.x;
    sdata[threadIdx.x] = 0;
    while (i < BLOCKS/4)
    {
        sdata[threadIdx.x] += g_idata[i] + g_idata[i+blockSize];
        i += gridSize;
    }
    __syncthreads();
    // do reduction in shared mem
    if (blockSize >= 512)
    {
        if (threadIdx.x < 256)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (threadIdx.x < 128)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (threadIdx.x < 64)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 64];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32)
        warpReduce<blockSize>(sdata, threadIdx.x);
    // write result for this block to global mem
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    return g_odata;
}
